#include "hip/hip_runtime.h"

#include "cuda_inpainting.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace cv;


const int CudaInpainting::RADIUS = 16;
const float CudaInpainting::RANGE_RATIO = 2.0f;

const int CudaInpainting::PATCH_WIDTH = CudaInpainting::RADIUS;
const int CudaInpainting::PATCH_HEIGHT = CudaInpainting::RADIUS;
const int CudaInpainting::NODE_WIDTH = CudaInpainting::PATCH_WIDTH / 2;
const int CudaInpainting::NODE_HEIGHT = CudaInpainting::PATCH_HEIGHT / 2;
const float CudaInpainting::CONST_FULL_MSG = CudaInpainting::PATCH_WIDTH * 
			CudaInpainting::PATCH_HEIGHT * 255 * 255 * 3 / 2.0f;

// a hepler fo copying memory to GPU
static void CopyToDevice(void *src, void *dst, uint32_t size) {
	hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

// a helper to copying memoery from GPU to the host
static void CopyFromDevice(void *src, void *dst, uint32_t size) {
	hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

// public functions


// the constructor
// take one arguement as the input image file
CudaInpainting::CudaInpainting(const char *path) {
	initFlag = false;
	image = imread(path, CV_LOAD_IMAGE_COLOR);
	imageData = nullptr;
	if(!image.data) {
		cout << "Image loading failed" << endl;
		return;
	}
	image.convertTo(image, CV_32FC3);
	
	// copy the image data to float array
	imageData = new float[3 * image.cols * image.rows];
	hipMalloc((void**)&deviceImageData, sizeof(float) * 3 * image.cols * image.rows);
	if(!imageData) {
		cout << "Memory allocation failed" << endl;
		hipFree(deviceImageData);
		return;
	}
	for(int y = 0; y < image.rows; ++y) {
		for(int x = 0; x < image.cols; ++x) {
			Vec3f vec = image.at<Vec3f>(y, x);
			imageData[3 * image.cols * y + 3 * x] = vec[0];
			imageData[3 * image.cols * y + 3 * x + 1] = vec[1];
			imageData[3 * image.cols * y + 3 * x + 2] = vec[2];
		}
	}

	// copy the raw data to the GPU
	CopyToDevice(imageData, deviceImageData, sizeof(float) * 3 * image.cols * image.rows);
	imgWidth = image.cols;
	imgHeight = image.rows;

	// initialize all the ointers
	choiceList = nullptr;
	nodeTable = nullptr;
	patchList = nullptr;
	
	devicePatchList = nullptr;
	deviceSSDTable = nullptr;
	deviceNodeTable = nullptr;
	deviceMsgTable = nullptr;
	deviceFillMsgTable = nullptr;
	deviceEdgeCostTable = nullptr;
	deviceChoiceList = nullptr;
}

// destructor for the CudaInpainting
CudaInpainting::~CudaInpainting() {
	if(imageData) {
		delete imageData;
		hipFree(deviceImageData);
	}
	if(choiceList)
		delete choiceList;
	if(patchList)
		delete patchList;
	if(nodeTable)
		delete nodeTable;

	if(devicePatchList)
		hipFree(devicePatchList);
	if(deviceNodeTable)
		hipFree(deviceNodeTable);
	if(deviceMsgTable)
		hipFree(deviceMsgTable);
	if(deviceFillMsgTable)
		hipFree(deviceFillMsgTable);
	if(deviceEdgeCostTable)
		hipFree(deviceEdgeCostTable);
	if(deviceChoiceList)
		hipFree(deviceChoiceList);
}

// GPU function to copy memory
__global__ void deviceCopyMem(float *src, float *dst, int elem) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x,
	    totalSize = blockDim.x * gridDim.x;
	for(int i = idx; i < elem; i += totalSize) {
		dst[i] = src[i];
		//printf("%d .  %f <= %f\n", i, dst[i], src[i]);
	}
}

// the main function for the image inpainting procedure
bool CudaInpainting::Inpainting(int x,int y, int width, int height, int iterTime) {
	Patch patch(x, y, width, height);
	// first generate the rounded up patch
	maskPatch = RoundUpArea(patch);

	// generate the candidate patches list
	GenPatches();
	hipDeviceSynchronize();
	if(patchListSize == 0)
		return true;

	// begin to calculate toe SSD table
	CalculateSSDTable();
	hipDeviceSynchronize();

	/*
	SSDEntry ent;
	int xx = 7, yy = 2;
	EPOS pos = LEFT_RIGHT;
	CopyFromDevice(deviceSSDTable+yy*patchListSize+xx, &ent, sizeof(SSDEntry));
	cout << "SSDEntry 2 -> 3 UP_DOWN: " << ent.data[pos] << endl;
	*/

	// before the iterations, we need to initialize the node table and the message
	InitNodeTable();
	deviceCopyMem<<<dim3(32,1), dim3(1024,1)>>>(deviceFillMsgTable, deviceMsgTable, nodeWidth * nodeHeight * DIR_COUNT * patchListSize);
	hipDeviceSynchronize();

	for(int i = 0; i < iterTime; i++) {
		RunIteration(i);
		deviceCopyMem<<<dim3(32,1), dim3(1024,1)>>>(deviceFillMsgTable, deviceMsgTable, nodeWidth * nodeHeight * DIR_COUNT * patchListSize);
		hipDeviceSynchronize();
		cout<<"ITERATION "<<i<<endl;
	}

	// calculate the best patch for each node
	SelectPatch();

	// fill the patch into the original image 
	FillPatch();

	// use a median filter to make the edge between two patches more smooth
	Rect rect(maskPatch.x - NODE_WIDTH, maskPatch.y - NODE_HEIGHT, 
			maskPatch.width + 2 * NODE_WIDTH, maskPatch.height + 2 * NODE_HEIGHT);
	Mat subMat = image(rect);
	Mat matArr[3];
	// split into multiple color channel
	split(subMat, matArr);
	
	for(int i = 0; i < 3; i++) {
		matArr[i].convertTo(matArr[i], CV_8U);
		medianBlur(matArr[i], matArr[i], 3);
		//GaussianBlur(matArr[i], matArr[i], Size(9,9), 0, 0);
		matArr[i].convertTo(matArr[i], CV_32F);
	}
	// merge the color changes into a color image
	merge(matArr, 3, subMat);

	return true;
}

Mat CudaInpainting::GetImage() {
	return image;
}

// private functions
CudaInpainting::Patch CudaInpainting::RoundUpArea(Patch p) {
	Patch res;
	res.x = (p.x / NODE_WIDTH) * NODE_WIDTH;
	res.y = (p.y / NODE_HEIGHT) * NODE_HEIGHT;
	res.width = (p.x + p.width +NODE_WIDTH - 1) / NODE_WIDTH * NODE_WIDTH - res.x;
	res.height = (p.y + p.height + NODE_WIDTH - 1) / NODE_HEIGHT * NODE_HEIGHT - res.y;
	return res;
}


// to judge if two given patches have overlap region
bool CudaInpainting::OverlapPatch(Patch& p1, Patch& p2) {
	int mLX = p1.x < p2.x ? p2.x : p1.x,
	    mRX = (p1.x+p1.width) < (p2.x+p2.width) ? (p1.x+p1.width) : (p2.x+p2.width),
	    mTY = p1.y < p2.y ? p2.y : p1.y,
	    mBY = (p1.y+p1.height) < (p2.y+p2.height) ? (p1.y+p1.height) : (p2.y+p2.height);
	return mRX > mLX && mBY > mTY;
}

// generate the patches list
void CudaInpainting::GenPatches() {
	vector<Patch> tmpPatchList;
	Patch p = maskPatch;
	cout << "x=" << p.x << " y=" << p.y << " width=" << p.width << " height=" << p.height << endl;
	int hh = image.rows / NODE_HEIGHT,
	    ww = image.cols / NODE_WIDTH;
	float midX = p.x + p.width / 2,
	      midY = p.y + p.height / 2;
	for(int i = 1; i <= hh; i++) {
		for(int j = 1; j <= ww; j++) {
			int cX, cY;
			float fcx = j * NODE_WIDTH, fcy = i * NODE_HEIGHT;
			cY = i * NODE_HEIGHT - NODE_HEIGHT;
			cX = j * NODE_WIDTH - NODE_WIDTH;
			if(!(fabsf(fcx - midX) * 2 / p.width < RANGE_RATIO && fabsf(fcy - midY) * 2 / p.height < RANGE_RATIO))
				continue;
			if(image.rows - cY < PATCH_HEIGHT || image.cols - cX < PATCH_WIDTH)
				continue;
			Patch cur(cX, cY, PATCH_WIDTH, PATCH_HEIGHT);
			if(!OverlapPatch(cur, p))
				tmpPatchList.push_back(cur);
		}
	}
	patchListSize = tmpPatchList.size();
	if(tmpPatchList.size() == 0)
		return;
	hipMalloc((void**)&devicePatchList, sizeof(Patch) * tmpPatchList.size());
	patchList = new Patch[tmpPatchList.size()];
	if(!patchList) {
		cout << "NULL patchList! exit"<< endl;
		exit(-1);
	}
	for(int i = 0; i < tmpPatchList.size(); i++) {
		patchList[i] = tmpPatchList[i];
	}
	// copy the generated patches list to the GPU global memory
	CopyToDevice(patchList, devicePatchList, sizeof(Patch) * tmpPatchList.size());
	cout << "GenPatch done, " << patchListSize << " patches generated" << endl;
	int idx = 23;
	cout << "Patch => " << idx << " : x=" << patchList[idx].x << " y=" << patchList[idx].y << endl;
	cout << "devicePatchList=" << devicePatchList << endl;
}

// a helper to get the message position in the message table
__device__ inline int getMsgIdx(int x, int y, CudaInpainting::EDIR dir, int l, int ww, int hh, int len) {
	return y * ww * CudaInpainting::EDIR::DIR_COUNT * len + x * CudaInpainting::EDIR::DIR_COUNT * len +
		dir * len + l;
}

// a helper to get the edge cost position in the message table
__device__ inline int getEdgeCostIdx(int x, int y, int l, int ww, int hh, int len) {
	return y * ww * len  + x * len + l;
}

// calculate the SSD table on GPU
__global__ void deviceCalculateSSDTable(float *dImg, int ww, int hh, CudaInpainting::Patch *pl, CudaInpainting::SSDEntry *dSSDTable) {
	int len = gridDim.x;
	const int patchSize = CudaInpainting::PATCH_HEIGHT * CudaInpainting::PATCH_WIDTH;	
	__shared__ float pixels[CudaInpainting::PATCH_HEIGHT][CudaInpainting::PATCH_WIDTH][3];
	for(int i = threadIdx.x; i < patchSize; i += blockDim.x) {
		int yy = i / CudaInpainting::PATCH_WIDTH, xx = i % CudaInpainting::PATCH_WIDTH;
		int iyy = pl[blockIdx.x].y + yy, ixx = pl[blockIdx.x].x + xx;
		pixels[yy][xx][0] = dImg[iyy * ww * 3 + ixx * 3];
		pixels[yy][xx][1] = dImg[iyy * ww * 3 + ixx * 3 + 1];
		pixels[yy][xx][2] = dImg[iyy * ww * 3 + ixx * 3 + 2];
	}

	__syncthreads();

	for(int i = threadIdx.x; i < len; i += blockDim.x) {
		int px = pl[i].x, py = pl[i].y;
		for(int j = 0; j < CudaInpainting::EPOS_COUNT; j++) {
			float res = 0;
			int WW, HH;
			int pxx, pyy;
			switch(j) {
				case CudaInpainting::UP_DOWN:
					WW = CudaInpainting::PATCH_WIDTH;
					HH = CudaInpainting::NODE_HEIGHT;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx;
							pyy = py + dy;
							float rr = pixels[dy + CudaInpainting::NODE_HEIGHT][dx][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy + CudaInpainting::NODE_HEIGHT][dx][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy + CudaInpainting::NODE_HEIGHT][dx][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
				case CudaInpainting::DOWN_UP:
					WW = CudaInpainting::PATCH_WIDTH;
					HH = CudaInpainting::NODE_HEIGHT;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx;
							pyy = py + dy + CudaInpainting::NODE_HEIGHT;
							float rr = pixels[dy][dx][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy][dx][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy][dx][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
				case CudaInpainting::RIGHT_LEFT:
					WW = CudaInpainting::NODE_WIDTH;
					HH = CudaInpainting::PATCH_HEIGHT;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx + CudaInpainting::NODE_WIDTH;
							pyy = py + dy;
							float rr = pixels[dy][dx][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy][dx][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy][dx][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
				case CudaInpainting::LEFT_RIGHT:
					WW = CudaInpainting::NODE_WIDTH;
					HH = CudaInpainting::PATCH_HEIGHT;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx;
							pyy = py + dy;
							float rr = pixels[dy][dx + CudaInpainting::NODE_WIDTH][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy][dx + CudaInpainting::NODE_WIDTH][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy][dx + CudaInpainting::NODE_WIDTH][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
			}
			dSSDTable[blockIdx.x * len + i].data[j] = res;
		}
	}
}

void CudaInpainting::CalculateSSDTable() {
	hipMalloc((void**)&deviceSSDTable, sizeof(SSDEntry) * patchListSize * patchListSize);
	if(devicePatchList && deviceSSDTable) {
		cout << "Calculate SSDTable" << endl;
		int len = PATCH_HEIGHT * PATCH_WIDTH;
		if(len > 1024)
			len = 1024;
		cout << "CUDA PARAM: " << patchListSize << "=>" << len << endl;
		deviceCalculateSSDTable<<<dim3(patchListSize, 1), dim3(len, 1)>>>(deviceImageData, imgWidth, imgHeight, devicePatchList, deviceSSDTable);
	}
}

__device__ float deviceCalculateSSD(float *dImg, int w, int h, CudaInpainting::Patch p1, CudaInpainting::Patch p2, CudaInpainting::EPOS pos) {
	float res = 0;
	int ww, hh;
	int p1x, p1y, p2x, p2y;
	switch(pos) {
		case CudaInpainting::UP_DOWN:
		case CudaInpainting::DOWN_UP:
			if(pos == CudaInpainting::UP_DOWN) {
				p1x = p1.x;
				p1y = p1.y;
				p2x = p2.x;
				p2y = p2.y;
			} else {
				p1x = p2.x;
				p1y = p2.y;
				p2x = p1.x;
				p2y = p1.y;
			}
			ww = CudaInpainting::PATCH_WIDTH;
			hh = CudaInpainting::NODE_HEIGHT;
			for(int i = 0; i < hh; ++i) {
				for(int j = 0; j < ww; ++j) {
					float rr = dImg[(p1y + CudaInpainting::NODE_HEIGHT + i) * w * 3 + (p1x + j) * 3] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3],
					      gg = dImg[(p1y + CudaInpainting::NODE_HEIGHT + i) * w * 3 + (p1x + j
) * 3 + 1] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 1],
					      bb = dImg[(p1y + CudaInpainting::NODE_HEIGHT + i) * w * 3 + (p1x + j
) * 3 + 2] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 2];
					rr *= rr;
					gg *= gg;
					bb *= bb;
					res += rr + gg + bb;
				}
			}
			break;
		case CudaInpainting::LEFT_RIGHT:
		case CudaInpainting::RIGHT_LEFT:
			if(pos == CudaInpainting::LEFT_RIGHT) {
				p1x = p1.x;
				p1y = p1.y;
				p2x = p2.x;
				p2y = p2.y;
			} else {
				p1x = p2.x;
				p1y = p2.y;
				p2x = p1.x;
				p2y = p1.y;
			}
			ww = CudaInpainting::NODE_WIDTH;
			hh = CudaInpainting::PATCH_HEIGHT;
			for(int i = 0; i < hh; ++i) {
				for(int j = 0; j < ww; ++j) {
					float rr = dImg[(p1y + i) * w * 3 + (p1x + CudaInpainting::NODE_WIDTH + j) * 3] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3],
					      gg = dImg[(p1y + i) * w * 3 + (p1x + CudaInpainting::NODE_WIDTH + j) * 3 + 1] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 1],
					      bb = dImg[(p1y + i) * w * 3 + (p1x + CudaInpainting::NODE_WIDTH + j) * 3 + 2] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 2];
					rr *= rr;
					gg *= gg;
					bb *= bb;
					res += rr + gg + bb;
				}
			}
			break;
	}
	return res;
}

// initialize the coordinates of node in table
__global__ void deviceInitFirst(CudaInpainting::Node* dNodeTable, CudaInpainting::Patch p) {
	int ww = gridDim.x;
	dNodeTable[ww * threadIdx.x + blockIdx.x].x = p.x + blockIdx.x * CudaInpainting::NODE_WIDTH;
	dNodeTable[ww * threadIdx.x + blockIdx.x].y = p.y + threadIdx.x * CudaInpainting::NODE_HEIGHT;
}

// the constructor of patch on GPU
__device__ CudaInpainting::Patch::Patch(int ww, int hh) {
	width = ww;
	height = hh;
}

// the initialize node table on GPU
__global__ void deviceInitNodeTable(float *dImg, int w, int h, CudaInpainting::Patch p, CudaInpainting::Node* dNodeTable, float *dMsgTable, float *dEdgeCostTable, CudaInpainting::Patch *dPatchList, int len) {
	int hh = gridDim.y, ww = gridDim.x;

	for(int i = threadIdx.x; i < len; i += blockDim.x * blockDim.y) {
		// initialize the message with the very large values
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_UP, i, ww, hh, len)] = CudaInpainting::CONST_FULL_MSG;
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_DOWN, i, ww, hh, len)] = CudaInpainting::CONST_FULL_MSG;
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_LEFT, i, ww, hh, len)] = CudaInpainting::CONST_FULL_MSG;
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_RIGHT, i, ww, hh, len)] = CudaInpainting::CONST_FULL_MSG;

		// initialize the edge cost 
		float val = 0;
		CudaInpainting::Patch curPatch(CudaInpainting::PATCH_WIDTH, CudaInpainting::PATCH_HEIGHT);
		
		// to judge if the current node is on the edge of the node table
		if(((blockIdx.y == 0 || blockIdx.y == hh - 1) && (/*blockIdx.x >= 0 && */blockIdx.x <= ww - 1 )) ||
					((blockIdx.x == 0 || blockIdx.x == ww - 1) && (/*blockIdx.y >= 0 && */blockIdx.y <= hh - 1))) {
			int nodeIdx = ww * blockIdx.y + blockIdx.x;
			int valCount = 0;
			if(blockIdx.x == 0) {
				curPatch.x = dNodeTable[nodeIdx].x - CudaInpainting::PATCH_WIDTH;
				curPatch.y = dNodeTable[nodeIdx].y - CudaInpainting::NODE_HEIGHT;
				val += deviceCalculateSSD(dImg, w, h, curPatch, dPatchList[i], CudaInpainting::LEFT_RIGHT);
				++valCount;
			} else {
				curPatch.x = dNodeTable[nodeIdx].x;
				curPatch.y = dNodeTable[nodeIdx].y - CudaInpainting::NODE_HEIGHT;
				val += deviceCalculateSSD(dImg, w, h, dPatchList[i], curPatch, CudaInpainting::LEFT_RIGHT);
				++valCount;
			}
			if(blockIdx.y == 0) {
				curPatch.x = dNodeTable[nodeIdx].x - CudaInpainting::NODE_WIDTH;
				curPatch.y = dNodeTable[nodeIdx].y - CudaInpainting::PATCH_HEIGHT;
				val += deviceCalculateSSD(dImg, w, h, curPatch, dPatchList[i], CudaInpainting::UP_DOWN);
				++valCount;
			} else {
				curPatch.x = dNodeTable[nodeIdx].x - CudaInpainting::NODE_WIDTH;
				curPatch.y = dNodeTable[nodeIdx].y;
				val += deviceCalculateSSD(dImg, w, h, dPatchList[i], curPatch, CudaInpainting::UP_DOWN);
				++valCount;
			}
			val /= valCount;
		}
		if(val < 0.5f) {
			val = CudaInpainting::CONST_FULL_MSG;
		}
		dEdgeCostTable[getEdgeCostIdx(blockIdx.x, blockIdx.y, i, ww, hh, len)] = val;
	}
	// just for debug
	/*
	__syncthreads();
	if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 1) {
		printf("(%d,%d) ", blockIdx.x, blockIdx.y);
		for(int i = 0; i < len; i++) {
			printf("%f ", dEdgeCostTable[getEdgeCostIdx(blockIdx.x, blockIdx.y, i, ww, hh, len)]);
		}
		printf("\n");
	}
	*/
}


// wrap for the initialization of the node table
void CudaInpainting::InitNodeTable() {
	nodeHeight = maskPatch.height / NODE_HEIGHT + 1;
	nodeWidth = maskPatch.width / NODE_WIDTH + 1;
	cout << "NodeTable => width=" << nodeWidth << " height=" << nodeHeight << endl;
	int totalElement = nodeWidth * nodeHeight * DIR_COUNT * patchListSize;
	cout << hipGetErrorString(hipMalloc((void**)&deviceNodeTable, sizeof(Node) * nodeWidth * nodeHeight)) << endl;
	cout << hipGetErrorString(hipMalloc((void**)&deviceMsgTable, sizeof(float) * totalElement)) << endl;
	cout << hipGetErrorString(hipMalloc((void**)&deviceFillMsgTable, sizeof(float) * totalElement)) << endl;
	cout << hipGetErrorString(hipMalloc((void**)&deviceEdgeCostTable, sizeof(float) * nodeWidth * nodeHeight * patchListSize)) << endl;
	if(deviceNodeTable && deviceMsgTable && deviceFillMsgTable && deviceEdgeCostTable) {
		cout << "Initialize the Node Table and Message Table" << endl;
		// initialize node table
		deviceInitFirst<<<dim3(nodeWidth, 1), dim3(nodeHeight,1)>>>(deviceNodeTable, maskPatch);

		// initialize the messages in the node table
		deviceInitNodeTable<<<dim3(nodeWidth, nodeHeight), dim3(512,1)>>>(deviceImageData, imgWidth, imgHeight, maskPatch, deviceNodeTable, deviceFillMsgTable, deviceEdgeCostTable, devicePatchList, patchListSize);
	} else {
		cout << " Failed to hipMalloc" << endl;
	}

	// initialize the node table on CPU
	nodeTable = new Node[nodeWidth * nodeHeight];
	if(nodeTable) {
		for(int i = 0; i < nodeHeight; ++i) {
			for(int j = 0; j < nodeWidth; ++j) {
				nodeTable[i * nodeWidth + j].x = maskPatch.x + j * NODE_WIDTH;
				nodeTable[i * nodeWidth + j].y = maskPatch.y + i * NODE_HEIGHT;
				/*
				printf("outside: (%d,%d) => (%d,%d)\n", j, i, nodeTable[i*nodeWidth+j].x,
						nodeTable[i*nodeHeight+j].y);
				*/
			}
		}
	}
}

// the iteration function which will be run on GPU
__global__ void deviceIteration(CudaInpainting::SSDEntry *dSSDTable, float *dEdgeCostTable, CudaInpainting::Patch *dPatchList, int len, float *dMsgTable, float *dFillMsgTable, int times) {
	int hh = gridDim.y, ww = gridDim.x, i = blockIdx.y, j = blockIdx.x;
	float aroundMsg, msgCount, matchFactor;
	float msgFactor = 0.8f;
	matchFactor = 1.2f;
	msgCount = msgFactor * 3 + matchFactor + 1;
	/*
	int bottom = hh - 1 - i, left = ww - 1 - j;
	if(times < i && times < j && times < bottom && times < left)
		return;
	*/
	// each thread handle one patch in all directions
	for(int ll = threadIdx.x; ll < len; ll += blockDim.x) {
		// use register to optimize the running time
		float up_val, down_val, left_val, right_val;
		// up
		if(i != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_UP, ll, ww, hh, len);
			up_val = dFillMsgTable[targetIdx];
		}
		// down
		if(i != hh - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_DOWN, ll, ww, hh, len);
			down_val = dFillMsgTable[targetIdx];
		}
		// left
		if(j != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_LEFT, ll, ww, hh, len);
			left_val = dFillMsgTable[targetIdx];
		}
		// right
		if(j != ww - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, ll, ww, hh, len);
			right_val = dFillMsgTable[targetIdx];
		}

		for(int k = 0; k < len; ++k) {
			/*
			float dX = (dPatchList[k].x - dPatchList[ll].x) / (float)CudaInpainting::NODE_WIDTH,
			      dY = dPatchList[k].y - dPatchList[ll].y / (float)CudaInpainting::NODE_HEIGHT;
			float distDiff = (dX * dX + dY * dY) * 3;
			*/
			float distDiff = 1;
			aroundMsg = 0;
			if(i != 0) {
				aroundMsg += dMsgTable[getMsgIdx(j, i - 1, CudaInpainting::DIR_DOWN, k, ww, hh, len)];
			} else {
				aroundMsg += CudaInpainting::CONST_FULL_MSG;
			}
			if(i != hh - 1) {
				aroundMsg += dMsgTable[getMsgIdx(j, i + 1, CudaInpainting::DIR_UP, k, ww, hh, len)];
			} else {
				aroundMsg += CudaInpainting::CONST_FULL_MSG;
			}
			if(j != 0) {
				aroundMsg += dMsgTable[getMsgIdx(j - 1, i, CudaInpainting::DIR_RIGHT, k, ww, hh, len)];
			} else {
				aroundMsg += CudaInpainting::CONST_FULL_MSG;
			}
			if(j != ww - 1) {
				aroundMsg += dMsgTable[getMsgIdx(j + 1, i, CudaInpainting::DIR_LEFT, k, ww, hh, len)];
			} else {
				aroundMsg += CudaInpainting::CONST_FULL_MSG;
			}
			aroundMsg *= msgFactor;
			float edgeVal = dEdgeCostTable[getEdgeCostIdx(j, i, k, ww, hh, len)];
			aroundMsg += edgeVal;
			float val, oldVal;
			// up
			if(i != 0) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::DOWN_UP] * matchFactor * distDiff;
				val -= dMsgTable[getMsgIdx(j, i - 1, CudaInpainting::DIR_DOWN, k, ww, hh, len)] * msgFactor;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_UP, ll, ww, hh, len);
				val /= msgCount;
				oldVal = up_val;
				//printf("(%d,%d,-%d) => val=%f\n", j, i, ll, val);
				if(val < oldVal) {
					up_val = val;
				}
			}
			// down
			if(i != hh - 1) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::UP_DOWN] * matchFactor * distDiff;
				val -= dMsgTable[getMsgIdx(j, i + 1, CudaInpainting::DIR_UP, k, ww, hh, len)] * msgFactor;
				val /= msgCount;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_DOWN, ll, ww, hh, len);
				oldVal = down_val;
				if(val < oldVal) {
					down_val = val;
				}
			}
			// left
			if(j != 0) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::RIGHT_LEFT] * matchFactor * distDiff;
				val -= dMsgTable[getMsgIdx(j - 1, i, CudaInpainting::DIR_RIGHT, k, ww, hh, len)] * msgFactor;
				val /= msgCount;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_LEFT, ll, ww, hh, len);
				oldVal = left_val;
				if(val < oldVal) {
					left_val = val;
				}
			}
			// right
			if(j != ww - 1) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::LEFT_RIGHT] * matchFactor;
				val -= dMsgTable[getMsgIdx(j + 1, i, CudaInpainting::DIR_LEFT, k, ww, hh, len)] * msgFactor;
				//printf("(%d,%d,-%d) => val=%f oldVal=%f\n", j, i, ll, val, oldVal);
				val /= msgCount;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, ll, ww, hh, len);
				oldVal = right_val;
				if(val < oldVal) {
					right_val = val;
				}
			}
		}
		// up
		if(i != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_UP, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = up_val;
		}
		// down
		if(i != hh - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_DOWN, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = down_val;
		}
		// left
		if(j != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_LEFT, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = left_val;
		}
		// right
		if(j != ww - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = right_val;
		}

		/*
		if(edgeVal > 0.5)
			++msgCount;
		*/
		//printf("msgCount=%f %f\n", msgCount, edgeVal);
		//printf("(%d,%d,%d) => aroundMsg=%f\n", j, i, k, aroundMsg);
		
	}

	/*
	if(blockIdx.x == 0 && blockIdx.y == 1 && threadIdx.x == 0) {
		printf("(%d,%d) %d\n", blockIdx.x, blockIdx.y, len);
		for(int k = 0; k < len; k++) {
			printf("%f ", dFillMsgTable[getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, k, ww,hh,len)]);
		}
		printf("\n");
	}
	*/
}

// the wrap function for iteration in Belief Propagation
void CudaInpainting::RunIteration(int times) {
	if(deviceMsgTable && deviceFillMsgTable && deviceSSDTable && deviceEdgeCostTable) {
		cout << "Run Iteration" << endl;
		int lim = 1024;
		if(patchListSize < lim) {
			lim = patchListSize;
		}
		deviceIteration<<<dim3(nodeWidth, nodeHeight),dim3(lim, 1)>>>(deviceSSDTable, deviceEdgeCostTable,devicePatchList, patchListSize, deviceMsgTable, deviceFillMsgTable, times);
	}
}


// select the best patch for each node on GPU
__global__ void deviceSelectPatch(float *dMsgTable, float *dEdgeCostTable, int *dChoiceList, 
		int ww, int hh,int len) {	
	int xx = blockDim.x * blockIdx.x + threadIdx.x, yy = blockDim.y * blockIdx. y + threadIdx.y;
	if(xx < ww && yy < hh) {
		float maxB = 0;
		int maxIdx = -1;
		for(int k = 0; k < len; ++k) {
			float bl = -dEdgeCostTable[getEdgeCostIdx(xx, yy, k, ww, hh, len)];
			float val;
			if(yy - 1 >= 0) {
				val = dMsgTable[getMsgIdx(xx, yy - 1, CudaInpainting::DIR_DOWN, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(yy + 1 < hh) {
				val = dMsgTable[getMsgIdx(xx, yy + 1, CudaInpainting::DIR_UP, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(xx - 1 >= 0) {
				val = dMsgTable[getMsgIdx(xx - 1, yy, CudaInpainting::DIR_RIGHT, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(xx + 1 < ww) {
				val = dMsgTable[getMsgIdx(xx + 1, yy, CudaInpainting::DIR_LEFT, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(bl > maxB || maxIdx < 0) {
				maxB = bl;
				maxIdx = k;
			}
		}
		//printf("(%d,%d) (%d,%d) => max %f %d\n", ww, hh, xx, yy, maxB, maxIdx);
		dChoiceList[yy * ww + xx] = maxIdx;
	}
}

// the wraper for selecting best patch on GPU
void CudaInpainting::SelectPatch() {
	choiceList = new int[nodeWidth * nodeHeight];
	hipMalloc((void**)&deviceChoiceList, sizeof(float) * nodeWidth * nodeHeight);
	if(choiceList && deviceChoiceList && deviceEdgeCostTable && deviceMsgTable) {
		cout << "Select the Best Patch" << endl;
		deviceSelectPatch<<<dim3((nodeWidth+15)/16, (nodeHeight+15)/16), dim3(16,16)>>>(deviceMsgTable, deviceEdgeCostTable, deviceChoiceList, nodeWidth, nodeHeight, patchListSize);
		CopyFromDevice(deviceChoiceList, choiceList, sizeof(int) * nodeWidth * nodeHeight);
	}
}

// the helper to paste the best patch to the specified node
void CudaInpainting::PastePatch(Mat& img, Node& n, Patch& p) {
	int xx = n.x - NODE_WIDTH / 2,
	    yy = n.y - NODE_HEIGHT / 2;
	for(int i = 0; i < p.height / 2; ++i) {
		for(int j = 0; j < p.width / 2; ++j) {
			img.at<Vec3f>(yy + i, xx + j) = img.at<Vec3f>(p.y + NODE_HEIGHT/2 + i, p.x + NODE_WIDTH/2 + j);
		}
	}
}

// paste best patch for all node
void CudaInpainting::FillPatch() {
	int hh = nodeHeight,
	    ww = nodeWidth;
	// just print the result
	for(int i = 0; i < hh; ++i) {
		for(int j = 0; j < ww; ++j) {
			cout<<choiceList[j + i * ww]<<" ";
		}
		cout<<endl;
	}
	for(int i = 0; i < hh; ++i) {
		for(int j = 0; j < ww; ++j) {
			int label = choiceList[j + i * ww];
			if(label >= 0) {
				PastePatch(image, nodeTable[j + i * ww], patchList[label]);
			}
		}
	}
}



